#include "hip/hip_runtime.h"
/*
 * DO NOT ALTER OR REMOVE COPYRIGHT NOTICES OR THIS HEADER.
 *
 * The contents of this file are subject to the terms of the
 * Common Development and Distribution License (the "License").
 * You may not use this file except in compliance with the License.
 *
 * You can read the License at http://opensource.org/licenses/CDDL-1.0
 * See the License for the specific language governing permissions
 * and limitations under the License.
 *
 * When distributing Covered Code, include this CDDL Header Notice in
 * each file and include the License file (licence.txt). If applicable,
 * add the following below this CDDL Header, with the fields enclosed
 * by brackets [] replaced by your own identifying information:
 * "Portions Copyright [year] [name of copyright owner]"
 *
 * The Original Software is AtmoSwing.
 * The Original Software was developed at the University of Lausanne.
 * All Rights Reserved.
 *
 */

/*
 * Portions Copyright 2014-2015 Pascal Horton, Terranum.
 * Portions Copyright 2019 Pascal Horton, University of Bern.
 */

// Disable some MSVC warnings
#ifdef _MSC_VER
#pragma warning( disable : 4244 ) // C4244: conversion from 'unsigned __int64' to 'unsigned int', possible loss of data
#pragma warning( disable : 4267 ) // C4267: conversion from 'size_t' to 'int', possible loss of data
#endif

#include "asProcessorCuda.cuh"
#include <stdio.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>

#define _TIME_CUDA false

// The number of threads per block should be a multiple of 32 threads, because this provides optimal computing
// efficiency and facilitates coalescing.
static const int maxBlockSize = 1024;

__device__
void warpReduce64(volatile float *shared, int tid)
{
    shared[tid] += shared[tid + 32];
    shared[tid] += shared[tid + 16];
    shared[tid] += shared[tid + 8];
    shared[tid] += shared[tid + 4];
    shared[tid] += shared[tid + 2];
    shared[tid] += shared[tid + 1];
}

__device__
void warpReduce32(volatile float *shared, int tid)
{
    shared[tid] += shared[tid + 16];
    shared[tid] += shared[tid + 8];
    shared[tid] += shared[tid + 4];
    shared[tid] += shared[tid + 2];
    shared[tid] += shared[tid + 1];
}

__global__
void processS1grads(int blockSize, long candNb, int ptsNb, const float *data, const long *idxTarg, const long *idxArch, float w, float *out)
{
    const long blockId = gridDim.x * gridDim.y * blockIdx.z + blockIdx.y * gridDim.x + blockIdx.x;

    if (blockId < candNb) {
        int bs = blockSize;
        int nPts = ptsNb;
        long iCand = blockId;
        int iPt = threadIdx.x;

        extern __shared__ float mem[];

        float *diff = mem;
        float *amax = &diff[bs];

        // Process differences and get abs max
        if (iPt < nPts) {

            // Lookup data value
            float xi = data[idxTarg[iCand] * nPts + iPt];
            float yi = data[idxArch[iCand] * nPts + iPt];

            float diffi = xi - yi;
            float amaxi = fabs(xi);
            if (fabs(yi) > amaxi) {
                amaxi = fabs(yi);
            }

            diff[iPt] = fabs(diffi);
            amax[iPt] = amaxi;
        } else {
            // Set rest of the block to 0
            diff[iPt] = 0;
            amax[iPt] = 0;
        }
        __syncthreads();

        // Process sum reduction
        for (unsigned int stride = bs / 2; stride > 32; stride /= 2) {
            if (iPt < stride) {
                diff[iPt] += diff[iPt + stride];
                amax[iPt] += amax[iPt + stride];
            }
            __syncthreads();
        }
        if (bs >= 64) {
            if (iPt < 32) {
                warpReduce64(diff, iPt);
                warpReduce64(amax, iPt);
            }
        } else {
            if (iPt < 16) {
                warpReduce32(diff, iPt);
                warpReduce32(amax, iPt);
            }
        }

        // Process final score
        if (iPt == 0) {
            float res = 0;

            if (amax[0] == 0) {
                res = 200;
            } else {
                res = 100.0f * (diff[0] / amax[0]);
            }
            *(out + iCand) += res * w;
        }
    }
}

bool asProcessorCuda::ProcessCriteria(std::vector<std::vector<float *>> &data, std::vector<int> &indicesTarg,
                                      std::vector<std::vector<int>> &indicesArch,
                                      std::vector<std::vector<float>> &resultingCriteria,
                                      std::vector<int> &nbCandidates, std::vector<int> &colsNb,
                                      std::vector<int> &rowsNb, std::vector<float> &weights,
                                      std::vector<CudaCriteria> &criteria)
{
    int ptorsNb = weights.size();

#if _TIME_CUDA
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0.0f;
#endif

    // Sizes
    long candNb = 0;
    std::vector<long> indexStart(nbCandidates.size() + 1);
    for (int i = 0; i < nbCandidates.size(); i++) {
        indexStart[i] = candNb;
        candNb += nbCandidates[i];
    }
    indexStart[nbCandidates.size()] = candNb;

    // Alloc space for indices
#if _TIME_CUDA
    hipEventRecord(start);
#endif
    long *hIdxTarg, *dIdxTarg;
    hIdxTarg = (long *)malloc(candNb * sizeof(long));
    checkCudaErrors(hipMalloc((void **)&dIdxTarg, candNb * sizeof(long)));
    long *hIdxArch, *dIdxArch;
    hIdxArch = (long *)malloc(candNb * sizeof(long));
    checkCudaErrors(hipMalloc((void **)&dIdxArch, candNb * sizeof(long)));
#if _TIME_CUDA
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time to allocate IdxTarg and IdxArch:    %f\n", milliseconds);
#endif

#if _TIME_CUDA
    hipEventRecord(start);
#endif
    for (int i = 0; i < indicesTarg.size(); i++) {
        for (int j = 0; j < nbCandidates[i]; j++) {
            hIdxArch[indexStart[i] + j] = indicesArch[i][j];
            hIdxTarg[indexStart[i] + j] = indicesTarg[i];
        }
    }
#if _TIME_CUDA
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time to initialize IdxTarg and IdxArch:  %f\n", milliseconds);
#endif

    // Copy to device
#if _TIME_CUDA
    hipEventRecord(start);
#endif
    checkCudaErrors(hipMemcpy(dIdxTarg, hIdxTarg, candNb * sizeof(long), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dIdxArch, hIdxArch, candNb * sizeof(long), hipMemcpyHostToDevice));
#if _TIME_CUDA
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time to copy IdxTarg and IdxArch:        %f\n", milliseconds);
#endif

    // Alloc space for results
#if _TIME_CUDA
    hipEventRecord(start);
#endif
    float *hRes, *dRes;
    hRes = (float *)malloc(candNb * sizeof(float));
    checkCudaErrors(hipMalloc((void **)&dRes, candNb * sizeof(float)));
#if _TIME_CUDA
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time to allocate dRes:                   %f\n", milliseconds);
#endif

    // Init resulting array to 0s
#if _TIME_CUDA
    hipEventRecord(start);
#endif
    checkCudaErrors(hipMemset(dRes, 0, candNb * sizeof(float)));
#if _TIME_CUDA
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time to memset dRes:                     %f\n", milliseconds);
#endif

    // Get max predictor size
    long maxDataSize = 0;
    for (int iPtor = 0; iPtor < ptorsNb; iPtor++) {
        int ptsNb = colsNb[iPtor] * rowsNb[iPtor];
        long dataSize = data[iPtor].size() * ptsNb;
        if (dataSize > maxDataSize) {
            maxDataSize = dataSize;
        }
    }

    // Alloc space for data
#if _TIME_CUDA
    hipEventRecord(start);
#endif
    float *hData, *dData;
    hData = (float *)malloc(maxDataSize * sizeof(float));
    checkCudaErrors(hipMalloc((void **)&dData, maxDataSize * sizeof(float)));
#if _TIME_CUDA
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time to allocate dData:                  %f\n", milliseconds);
#endif

    // Loop over all predictors
    for (int iPtor = 0; iPtor < ptorsNb; iPtor++) {

        int ptsNb = colsNb[iPtor] * rowsNb[iPtor];
        float weight = weights[iPtor];
        long dataSize = data[iPtor].size() * ptsNb;

        // Copy data in the new arrays
#if _TIME_CUDA
        hipEventRecord(start);
#endif
        for (int iDay = 0; iDay < data[iPtor].size(); iDay++) {
            for (int iPt = 0; iPt < ptsNb; iPt++) {
                hData[iDay * ptsNb + iPt] = data[iPtor][iDay][iPt];
            }
        }
#if _TIME_CUDA
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("time to initialize hData:                %f\n", milliseconds);
#endif

        // Copy the data to the device
#if _TIME_CUDA
        hipEventRecord(start);
#endif
        checkCudaErrors(hipMemcpy(dData, hData, dataSize * sizeof(float), hipMemcpyHostToDevice));
#if _TIME_CUDA
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("time to copy data:                       %f\n", milliseconds);
#endif

        // Reduction only allowed on 1 block yet
        if (ptsNb > maxBlockSize) {
            printf("Using more than 1 gpu block (too much data points)\n");
            return false;
        }

        // Define block size (must be multiple of 32) and blocks nb
        int blockSize = (int)ceil(ptsNb / 32.0) * 32;
        int blocksNbXY = ceil(std::cbrt(candNb));
        int blocksNbZ = ceil((double)candNb / (blocksNbXY * blocksNbXY));
        dim3 blocksNb3D(blocksNbXY, blocksNbXY, blocksNbZ);

        // Launch kernel
#if _TIME_CUDA
        hipEventRecord(start);
#endif
        switch (criteria[iPtor]) {
            case S1grads:
                // 3rd <<< >>> argument is for the dynamically allocated shared memory
                processS1grads<<<blocksNb3D, blockSize, 2*blockSize*sizeof(float)>>>(blockSize, candNb, ptsNb, dData, dIdxTarg, dIdxArch, weight, dRes);
                break;
            default:
                printf("Criteria not yet implemented on GPU.");
                return false;
        }

        // Check for any errors launching the kernel
        checkCudaErrors(hipGetLastError());

        checkCudaErrors(hipDeviceSynchronize());
#if _TIME_CUDA
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("time for kernel:                         %f\n", milliseconds);
#endif
    }

    // Copy the resulting array to the device
#if _TIME_CUDA
    hipEventRecord(start);
#endif
    checkCudaErrors(hipMemcpy(hRes, dRes, candNb * sizeof(float), hipMemcpyDeviceToHost));
#if _TIME_CUDA
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time to copy results:                    %f\n", milliseconds);
#endif

    // Set the criteria values in the vector container
    for (int i = 0; i < nbCandidates.size(); i++) {
        std::vector<float> tmpCrit(nbCandidates[i]);

        for (int j = 0; j < nbCandidates[i]; j++) {
            tmpCrit[j] = hRes[indexStart[i] + j];
        }
        resultingCriteria[i] = tmpCrit;
    }

    free(hData);
    checkCudaErrors(hipFree(dData));
    free(hRes);
    checkCudaErrors(hipFree(dRes));
    free(hIdxTarg);
    checkCudaErrors(hipFree(dIdxTarg));
    free(hIdxArch);
    checkCudaErrors(hipFree(dIdxArch));

    return true;
}

bool asProcessorCuda::SelectBestDevice()
{
    hipError_t cudaStatus;
    bool showDeviceName = false;

    // Count the devices
    int devicesCount = 0;
    cudaStatus = hipGetDeviceCount(&devicesCount);
    if (cudaStatus != hipSuccess) {
        if (cudaStatus == hipErrorNoDevice) {
            printf("hipGetDeviceCount failed! Do you have a CUDA-capable GPU installed?\n");
            return false;
        } else if (cudaStatus == hipErrorInsufficientDriver) {
            printf("hipGetDeviceCount failed! No driver can be loaded to determine if any device exists.\n");
            return false;
        }

        printf("hipGetDeviceCount failed! Do you have a CUDA-capable GPU installed?\n");
        return false;
    }

    // Get some info on the devices
    int bestDevice = 0;
    int memSize = 0;
    struct hipDeviceProp_t deviceProps;
    for (int i_dev = 0; i_dev < devicesCount; i_dev++) {
        checkCudaErrors(hipGetDeviceProperties(&deviceProps, i_dev));
        if (showDeviceName) {
            printf("CUDA device [%s]\n", deviceProps.name);
        }

        // Compare memory
        if (deviceProps.totalGlobalMem > memSize) {
            memSize = deviceProps.totalGlobalMem;
            bestDevice = i_dev;
        }
    }

    // Select the best device
    checkCudaErrors(hipSetDevice(bestDevice));

    return true;
}

float *asProcessorCuda::MallocCudaData(int n)
{
    float *data;
    checkCudaErrors(hipMallocManaged(&data, n * sizeof(float)));

    return data;
}

void asProcessorCuda::FreeCudaData(float *data)
{
    checkCudaErrors(hipFree(data));
}

void asProcessorCuda::DeviceSynchronize()
{
    checkCudaErrors(hipDeviceSynchronize());
}

void asProcessorCuda::DeviceReset()
{
    hipDeviceReset();
}

