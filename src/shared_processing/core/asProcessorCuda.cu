#include "hip/hip_runtime.h"
/*
 * DO NOT ALTER OR REMOVE COPYRIGHT NOTICES OR THIS HEADER.
 *
 * The contents of this file are subject to the terms of the
 * Common Development and Distribution License (the "License").
 * You may not use this file except in compliance with the License.
 *
 * You can read the License at http://opensource.org/licenses/CDDL-1.0
 * See the License for the specific language governing permissions
 * and limitations under the License.
 *
 * When distributing Covered Code, include this CDDL Header Notice in
 * each file and include the License file (licence.txt). If applicable,
 * add the following below this CDDL Header, with the fields enclosed
 * by brackets [] replaced by your own identifying information:
 * "Portions Copyright [year] [name of copyright owner]"
 *
 * The Original Software is AtmoSwing.
 * The Original Software was developed at the University of Lausanne.
 * All Rights Reserved.
 *
 */

/*
 * Portions Copyright 2014-2015 Pascal Horton, Terranum.
 * Portions Copyright 2019 Pascal Horton, University of Bern.
 */

// Disable some MSVC warnings
#ifdef _MSC_VER
#pragma warning( disable : 4244 ) // C4244: conversion from 'unsigned __int64' to 'unsigned int', possible loss of data
#pragma warning( disable : 4267 ) // C4267: conversion from 'size_t' to 'int', possible loss of data
#endif

#include "asProcessorCuda.cuh"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>

// The number of threads per block should be a multiple of 32 threads, because this provides optimal computing
// efficiency and facilitates coalescing.
static const int blockSize = 1024;


// From https://riptutorial.com/cuda/example/22456/single-block-parallel-reduction-for-commutative-operator
__global__
void sumSingleBlock(int n, const float *a, float *out)
{
    int idx = threadIdx.x;
    float sum = 0;
    for (int i = idx; i < n; i += blockSize)
        sum += a[i];
    __shared__ float r[blockSize];
    r[idx] = sum;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) {
        if (idx < size)
            r[idx] += r[idx + size];
        __syncthreads();
    }
    if (idx == 0)
        *out = r[0];
}

// From https://devblogs.nvidia.com/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
__global__
void diff(int n, const float *x, const float *y, float *r)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < n; i += stride) {
        r[i] = x[i] - y[i];
    }
}

__global__
void maxAbs(int n, const float *x, const float *y, float *r)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < n; i += stride) {
        r[i] = fabs(x[i]);
        if (fabs(y[i]) > r[i]) {
            r[i] = fabs(y[i]);
        }
    }
}

__global__
void criteriaS1grads(int n, const float *x, const float *y, float *out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    __shared__ float diff[blockSize];
    __shared__ float amax[blockSize];

    for (int i = idx; i < n; i += stride) {
        float xi = x[i];
        float yi = y[i];

        float diffi = xi - yi;
        float amaxi = fabs(xi);
        if (fabs(yi) > amaxi) {
            amaxi = fabs(yi);
        }

        diff[i] = diffi;
        amax[i] = amaxi;
    }
    __syncthreads();

    float sumDiff = 0;
    float sumMax = 0;
    for (int i = idx; i < n; i += blockSize) {
        sumDiff += fabs(diff[i]);
        sumMax += amax[i];
    }

    __shared__ float rDiff[blockSize];
    __shared__ float rMax[blockSize];
    rDiff[idx] = sumDiff;
    rMax[idx] = sumMax;
    __syncthreads();

    for (int size = blockSize / 2; size > 0; size /= 2) {
        if (idx < size) {
            rDiff[idx] +=rDiff[idx + size];
            rMax[idx] +=rMax[idx + size];
        }
        __syncthreads();
    }
    if (idx == 0) {
        *out = 100.0f * (rDiff[0] / rMax[0]);

        if (rMax[0] == 0) {
            if (rDiff[0] == 0) {
                *out = 0;
            } else {
                *out = 200;
            }
        }
    }
}

bool asProcessorCuda::SelectBestDevice()
{
    hipError_t cudaStatus;
    bool showDeviceName = false;

    // Count the devices
    int devicesCount = 0;
    cudaStatus = hipGetDeviceCount(&devicesCount);
    if (cudaStatus != hipSuccess) {
        if (cudaStatus == hipErrorNoDevice) {
            printf("hipGetDeviceCount failed! Do you have a CUDA-capable GPU installed?\n");
            return false;
        } else if (cudaStatus == hipErrorInsufficientDriver) {
            printf("hipGetDeviceCount failed! No driver can be loaded to determine if any device exists.\n");
            return false;
        }

        printf("hipGetDeviceCount failed! Do you have a CUDA-capable GPU installed?\n");
        return false;
    }

    // Get some info on the devices
    int bestDevice = 0;
    int memSize = 0;
    struct hipDeviceProp_t deviceProps;
    for (int i_dev = 0; i_dev < devicesCount; i_dev++) {
        checkCudaErrors(hipGetDeviceProperties(&deviceProps, i_dev));
        if (showDeviceName) {
            printf("CUDA device [%s]\n", deviceProps.name);
        }

        // Compare memory
        if (deviceProps.totalGlobalMem > memSize) {
            memSize = deviceProps.totalGlobalMem;
            bestDevice = i_dev;
        }
    }

    // Select the best device
    checkCudaErrors(hipSetDevice(bestDevice));

    return true;
}

float *asProcessorCuda::MallocCudaData(int n)
{
    float *data;
    checkCudaErrors(hipMallocManaged(&data, n * sizeof(float)));

    return data;
}

void asProcessorCuda::FreeCudaData(float *data)
{
    checkCudaErrors(hipFree(data));
}

void asProcessorCuda::DeviceSynchronize()
{
    checkCudaErrors(hipDeviceSynchronize());
}

void asProcessorCuda::DeviceReset()
{
    hipDeviceReset();
}

bool asProcessorCuda::ProcessS1grads(float *out, const float *refData, const float *evalData, int rowsNb, int colsNb)
{
    int n = rowsNb * colsNb;

 

    int blocksNb = (n + blockSize - 1) / blockSize;

    if (blocksNb > 1) {
        printf("blocksNb > 1\n");
        return false;
    }


    bool m_checkNaNs = false;


    // Note here that the actual gradient data do not fill the entire data blocks,
    // but the rest being 0-filled, we can simplify the sum calculation !

    if (!m_checkNaNs) {
        criteriaS1grads<<<blocksNb, blockSize>>>(n, refData, evalData, out);
    } else {
        /*
        a2f refDataCorr = (!evalData.isNaN() && !refData.isNaN()).select(refData, 0);
        a2f evalDataCorr = (!evalData.isNaN() && !refData.isNaN()).select(evalData, 0);

        dividend = ((refDataCorr - evalDataCorr).abs()).sum();
        divisor = (refDataCorr.abs().max(evalDataCorr.abs())).sum();*/
    }





    return true;
}


__global__
void gpuPredictorCriteriaS1grads(float *criteria, const float *data, const int *indicesTarg,
                                 const int *indicesArch, const int *indexStart,
                                 const cudaPredictorsDataPropStruct dataProp, const int n_targ,
                                 const int n_cand, const int offset)
{

#if USE_STREAMS
    int i_cand = offset + threadIdx.x + blockIdx.x * blockDim.x;
    if (i_cand < n_cand) {
        // Find the target index
        float meanNbCand = float(n_cand) / float(n_targ);
        int i_targ = floorf(float(i_cand) / meanNbCand);

        if (i_targ < 0) {
            i_targ = 0;
        }

        if (i_targ >= n_targ) {
            i_targ = n_targ - 1;
        }

        // Check and correct
        if (i_cand < indexStart[i_targ]) {
            while (i_cand < indexStart[i_targ]) {
                i_targ--;

                if (i_targ < 0) {
                    printf("Device error: The target index is < 0 : i_targ = %d.\n", i_targ);
                    criteria[i_cand] = -9999;
                    return;
                }
            }
        }
        if (i_cand >= indexStart[i_targ + 1]) // safe
        {
            while (i_cand >= indexStart[i_targ + 1]) {
                i_targ++;

                if (i_targ >= n_targ) {
                    printf("Device error: The target index is >= n_targ : i_targ = %d (n_targ = %d)\n", i_targ, n_targ);
                    criteria[i_cand] = -9999;
                    return;
                }
            }
        }

        float criterion = 0;

        int targIndexBase = indicesTarg[i_targ] * dataProp.totPtsNb;
        int archIndexBase = indicesArch[i_cand] * dataProp.totPtsNb;

        for (int iPtor = 0; iPtor < dataProp.ptorsNb; iPtor++) {
            float dividend = 0, divisor = 0;
            int targIndex = targIndexBase + dataProp.indexStart[iPtor];
            int archIndex = archIndexBase + dataProp.indexStart[iPtor];

            for (int i = 0; i < dataProp.ptsNb[iPtor]; i++) {
                dividend += fabsf(data[targIndex] - data[archIndex]);
                divisor += fmaxf(fabsf(data[targIndex]), fabsf(data[archIndex]));

                targIndex++;
                archIndex++;
            }

            criterion += dataProp.weights[iPtor] * 100.0f * (dividend / divisor);
        }

        criteria[i_cand] = criterion;
    }

#else

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i_cand = index; i_cand < n_cand; i_cand += stride) {

        // Find the target index
        float meanNbCand = float(n_cand) / float(n_targ);
        int i_targ = (int)floorf(float(i_cand) / meanNbCand);

        if (i_targ < 0) {
            i_targ = 0;
        }

        if (i_targ >= n_targ) {
            i_targ = n_targ - 1;
        }

        // Check and correct
        if (i_cand < indexStart[i_targ]) {
            while (i_cand < indexStart[i_targ]) {
                i_targ--;
                if (i_targ < 0) {
                    printf("Device error: The target index is < 0 : i_targ = %d.\n", i_targ);
                    criteria[i_cand] = 9999;
                    return;
                }
            }
        }
        if (i_cand >= indexStart[i_targ + 1]) // safe
        {
            while (i_cand >= indexStart[i_targ + 1]) {
                i_targ++;
                if (i_targ >= n_targ) {
                    printf("Device error: The target index is >= n_targ : i_targ = %d (n_targ = %d)\n", i_targ, n_targ);
                    criteria[i_cand] = 9999;
                    return;
                }
            }
        }

        float criterion = 0;
        int targIndexBase = indicesTarg[i_targ] * dataProp.totPtsNb;
        int archIndexBase = indicesArch[i_cand] * dataProp.totPtsNb;

        for (int iPtor = 0; iPtor < dataProp.ptorsNb; iPtor++) {
            float dividend = 0, divisor = 0;
            int targIndex = targIndexBase + dataProp.indexStart[iPtor];
            int archIndex = archIndexBase + dataProp.indexStart[iPtor];

            for (int i = 0; i < dataProp.ptsNb[iPtor]; i++) {
                dividend += fabsf(data[targIndex] - data[archIndex]);
                divisor += fmaxf(fabsf(data[targIndex]), fabsf(data[archIndex]));

                targIndex++;
                archIndex++;
            }

            criterion += dataProp.weights[iPtor] * 100.0f * (dividend / divisor);
        }

        criteria[i_cand] = criterion;
    }

#endif
}


bool asProcessorCuda::ProcessCriteria(std::vector <std::vector<float *>> &data,
                                      std::vector<int> &indicesTarg,
                                      std::vector <std::vector<int>> &indicesArch,
                                      std::vector <std::vector<float>> &resultingCriteria,
                                      std::vector<int> &nbArchCandidates,
                                      std::vector<int> &colsNb, std::vector<int> &rowsNb,
                                      std::vector<float> &weights)
{

    // Get the data structure
    cudaPredictorsDataPropStruct struc;
    struc.ptorsNb = (int) weights.size();
    if (struc.ptorsNb > STRUCT_MAX_SIZE) {
        printf("The number of predictors is > %d. Please adapt the source code in asProcessorCuda::ProcessCriteria.\n",
               STRUCT_MAX_SIZE);
        return false;
    }

    struc.totPtsNb = 0;

    for (int iPtor = 0; iPtor < struc.ptorsNb; iPtor++) {
        struc.rowsNb[iPtor] = rowsNb[iPtor];
        struc.colsNb[iPtor] = colsNb[iPtor];
        struc.weights[iPtor] = weights[iPtor];
        struc.ptsNb[iPtor] = colsNb[iPtor] * rowsNb[iPtor];
        struc.indexStart[iPtor] = struc.totPtsNb;
        struc.totPtsNb += colsNb[iPtor] * rowsNb[iPtor];
    }

    // Sizes
    int nbArchCandidatesSum = 0;
    std::vector<int> indexStart(nbArchCandidates.size() + 1);
    for (int i = 0; i < nbArchCandidates.size(); i++) {
        indexStart[i] = nbArchCandidatesSum;
        nbArchCandidatesSum += nbArchCandidates[i];
    }
    indexStart[nbArchCandidates.size()] = nbArchCandidatesSum;

    // Blocks of threads
    int n_targ = nbArchCandidates.size();
    int n_cand = nbArchCandidatesSum;
    // The number of threads per block should be a multiple of 32 threads, because this provides optimal computing efficiency and facilitates coalescing.
    const int threadsPerBlock = 512; // no need to change
    int blocksNb = (n_cand + threadsPerBlock - 1) / threadsPerBlock;


#if USE_STREAMS
    // Create streams
    const int nStreams = 4; // no need to change
    // rowsNbPerStream must be dividable by nStreams and threadsPerBlock
    int rowsNbPerStream = ceil(float(nbArchCandidatesSum) / float(nStreams * threadsPerBlock)) * threadsPerBlock;
    // Streams
    hipStream_t stream[nStreams];
    for (int i = 0; i < nStreams; i++) {
        hipStreamCreate(&stream[i]);
    }
#endif

    // Data pointers
    float *arrData, *arrCriteria;
    int *arrIndicesTarg, *arrIndicesArch, *arrIndexStart;

    // Alloc space for data
    checkCudaErrors(hipMallocManaged(&arrData, data.size() * struc.totPtsNb * sizeof(float)));
    checkCudaErrors(hipMallocManaged(&arrCriteria, nbArchCandidatesSum * sizeof(float)));
    checkCudaErrors(hipMallocManaged(&arrIndicesTarg, nbArchCandidates.size() * sizeof(int)));
    checkCudaErrors(hipMallocManaged(&arrIndicesArch, nbArchCandidatesSum * sizeof(int)));
    checkCudaErrors(hipMallocManaged(&arrIndexStart, (nbArchCandidates.size() + 1) * sizeof(int)));

    // Copy data in the new arrays
    for (int iDay = 0; iDay < data.size(); iDay++) {
        for (int iPtor = 0; iPtor < struc.ptorsNb; iPtor++) {
            for (int iPt = 0; iPt < struc.ptsNb[iPtor]; iPt++) {
                arrData[iDay * struc.totPtsNb + struc.indexStart[iPtor] + iPt] = data[iDay][iPtor][iPt];
            }
        }
    }

    for (int i = 0; i < nbArchCandidates.size(); i++) {
        for (int j = 0; j < nbArchCandidates[i]; j++) {
            arrIndicesArch[indexStart[i] + j] = indicesArch[i][j];
        }
    }

    for (int i = 0; i < indicesTarg.size(); i++) {
        arrIndicesTarg[i] = indicesTarg[i];
    }

    for (int i = 0; i < indexStart.size(); i++) {
        arrIndexStart[i] = indexStart[i];
    }

    // Launch kernel on GPU
#if USE_STREAMS
    for (int i = 0; i < nStreams; i++) {
        int offset = i * rowsNbPerStream;
        blocksNb = rowsNbPerStream / threadsPerBlock;
        gpuPredictorCriteriaS1grads<<<blocksNb, threadsPerBlock, 0, stream[i]>>>(arrCriteria, arrData, arrIndicesTarg, arrIndicesArch, arrIndexStart, struc, n_targ, n_cand, offset);
    }
#else
    gpuPredictorCriteriaS1grads<<<blocksNb, threadsPerBlock>>>(arrCriteria, arrData, arrIndicesTarg, arrIndicesArch, arrIndexStart, struc, n_targ, n_cand, 0);
#endif

    // Check for any errors launching the kernel
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Set the criteria values in the vector container
    for (int i = 0; i < nbArchCandidates.size(); i++) {
        std::vector<float> tmpCrit(nbArchCandidates[i]);

        for (int j = 0; j < nbArchCandidates[i]; j++) {
            tmpCrit[j] = arrCriteria[indexStart[i] + j];
        }
        resultingCriteria[i] = tmpCrit;
    }

    // Cleanup

#if USE_STREAMS
    for (int i = 0; i< nStreams; i++) {
        hipStreamDestroy(stream[i]);
    }
#endif

    hipFree(arrData);
    hipFree(arrCriteria);
    hipFree(arrIndicesTarg);
    hipFree(arrIndicesArch);
    hipFree(arrIndexStart);

    return false;
}

