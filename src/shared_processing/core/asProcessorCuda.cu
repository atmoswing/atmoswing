#include "hip/hip_runtime.h"
/*
 * DO NOT ALTER OR REMOVE COPYRIGHT NOTICES OR THIS HEADER.
 *
 * The contents of this file are subject to the terms of the
 * Common Development and Distribution License (the "License").
 * You may not use this file except in compliance with the License.
 *
 * You can read the License at http://opensource.org/licenses/CDDL-1.0
 * See the License for the specific language governing permissions
 * and limitations under the License.
 *
 * When distributing Covered Code, include this CDDL Header Notice in
 * each file and include the License file (licence.txt). If applicable,
 * add the following below this CDDL Header, with the fields enclosed
 * by brackets [] replaced by your own identifying information:
 * "Portions Copyright [year] [name of copyright owner]"
 *
 * The Original Software is AtmoSwing. The Initial Developer of the
 * Original Software is Pascal Horton of the University of Lausanne.
 * All Rights Reserved.
 *
 */

/*
 * Portions Copyright 2014 Pascal Horton, Terr@num.
 */

// Disable some MSVC warnings
#ifdef _MSC_VER
    #pragma warning( disable : 4244 ) // C4244: conversion from 'unsigned __int64' to 'unsigned int', possible loss of data
    #pragma warning( disable : 4267 ) // C4267: conversion from 'size_t' to 'int', possible loss of data
#endif


#include "asProcessorCuda.cuh"

#include <stdio.h>
#include <time.h>

#define USE_THRUST 1
#define DO_PROFILE 1

#if USE_THRUST
    #include <thrust/host_vector.h>
    #include <thrust/device_vector.h>
    #include <thrust/transform.h>
    #include <thrust/for_each.h>
    #include <thrust/fill.h>
    #include <thrust/iterator/zip_iterator.h>
#else // USE_THRUST
    #include <hip/hip_runtime.h>
    #include <hip/hip_runtime.h>
    #include <>
#endif // USE_THRUST


#if USE_THRUST

struct gpuPredictorCriteriaS1grads
{
    template <typename Tuple>
    __host__ __device__
    void operator()(Tuple t)
    {
        // 0: targData, 1: archData, 2: dividend, 3: divisor
        // Dividend
        thrust::get<2>(t) = abs(thrust::get<0>(t)-thrust::get<1>(t));
        // Divisor
        thrust::get<3>(t) = thrust::max(abs(thrust::get<0>(t)), abs(thrust::get<1>(t)));
    }
};

struct gpuAddToCriteriaS1grads
{
    const float weight;
    gpuAddToCriteriaS1grads(float _weight) : weight(_weight) {}

    template <typename Tuple>
    __host__ __device__
    void operator()(Tuple t)
    {
        // 0: reducedDividend, 1: reducedDivisor, 2: resultingCriteria
        thrust::get<2>(t) += weight*100.0f*(thrust::get<0>(t)/thrust::get<1>(t));
    }
};

#else // USE_THRUST

__global__ void gpuPredictorCriteriaS1grads(float *criteria,
                                            const float *targData,
                                            const float *archData,
                                            const cudaPredictorsMetaDataStruct metaData,
                                            int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
    {
        criteria[index] = 0;

        for (int i_ptor=0; i_ptor<metaData.ptorsNb; i_ptor++)
        {
            float dividend = 0, divisor = 0;

            for (int i=0; i<metaData.rowsNb[i_ptor]; i++)
            {
                for (int j=0; j<metaData.colsNb[i_ptor]; j++)
                {
                    int subindex = metaData.indexStart[i_ptor]+i*metaData.colsNb[i_ptor]+j;

                    dividend += abs(targData[subindex]-archData[index*metaData.totPtsNb+subindex]);
                    divisor += max(abs(targData[subindex]),abs(archData[index*metaData.totPtsNb+subindex]));
                }
            }

            criteria[index] += metaData.weights[i_ptor]*100.0f*(dividend/divisor);
        }
    }
}

#endif // USE_THRUST

bool asProcessorCuda::ProcessCriteria(std::vector < float* > &vpTargData,
                                      std::vector < std::vector < float* > > &vvpArchData,
                                      std::vector < float > &criteriaValues,
                                      int size,
                                      std::vector < int > &colsNb,
                                      std::vector < int > &rowsNb,
                                      std::vector < float > &weights)
{

    #if USE_THRUST
    
    #if DO_PROFILE
        clock_t start, stop;
        float time;
        start = clock();
    #endif //DO_PROFILE

    // Allocate storage
    thrust::device_vector<float> resultingCriteria(size, 0);
    thrust::device_vector<float> reducedDivisor(size);
    thrust::device_vector<float> reducedDividend(size);
    thrust::device_vector<int> reducedKeys(size);

    #if DO_PROFILE
        stop = clock();   
        time = (float)(stop-start)/CLOCKS_PER_SEC*1000;
        fprintf(stderr, "First storage allocation: %f ms\n", time);
    #endif //DO_PROFILE

    // Number of predictors
    int ptorsNb = (int)weights.size();

    // Loop over every predictor
    for (int i_ptor=0; i_ptor<ptorsNb; i_ptor++)
    {
        // Number of points
        int ptsNb = colsNb[i_ptor]*rowsNb[i_ptor];

        #if DO_PROFILE
            start = clock();
        #endif //DO_PROFILE

        // Allocate storage
        thrust::host_vector<float> hostTargData(size*ptsNb);
        thrust::host_vector<float> hostArchData(size*ptsNb);
        thrust::device_vector<float> devTargData(size*ptsNb);
        thrust::device_vector<float> devArchData(size*ptsNb);
        thrust::device_vector<float> devDividend(size*ptsNb);
        thrust::device_vector<float> devDivisor(size*ptsNb);
        thrust::device_vector<int> keys(size*ptsNb);

        #if DO_PROFILE
            stop = clock();   
            time = (float)(stop-start)/CLOCKS_PER_SEC*1000;
            fprintf(stderr, "Predictor storage allocation: %f ms\n", time);

            start = clock();
        #endif //DO_PROFILE

        // Populate host vectors (to do only 1 copy to the device)
        for (int i_day=0; i_day<size; i_day++)
        {
            int destinationIndex = i_day*ptsNb;
            thrust::copy(vpTargData[i_ptor], vpTargData[i_ptor]+ptsNb, hostTargData.begin()+destinationIndex);
            thrust::copy(vvpArchData[i_day][i_ptor], vvpArchData[i_day][i_ptor]+ptsNb, hostArchData.begin()+destinationIndex);
            thrust::fill(keys.begin()+destinationIndex, keys.begin()+destinationIndex+ptsNb, i_day);
        }

        // Copy data to device
        devTargData = hostTargData;
        devArchData = hostArchData;

        #if DO_PROFILE
            stop = clock();   
            time = (float)(stop-start)/CLOCKS_PER_SEC*1000;
            fprintf(stderr, "Data copy: %f ms\n", time);

            start = clock();
        #endif //DO_PROFILE

        // Process dividend and divisor
        thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(devTargData.begin(), devArchData.begin(), devDividend.begin(), devDivisor.begin())),
                         thrust::make_zip_iterator(thrust::make_tuple(devTargData.end(), devArchData.end(), devDividend.end(), devDivisor.end())),
                         gpuPredictorCriteriaS1grads());

        #if DO_PROFILE
            stop = clock();   
            time = (float)(stop-start)/CLOCKS_PER_SEC*1000;
            fprintf(stderr, "Dividend and divisor calculation: %f ms\n", time);

            start = clock();
        #endif //DO_PROFILE

        // Proceed to reduction
        /*
        for (int i_day=0; i_day<size; i_day++)
        {
            int indexStart = i_day*ptsNb;
            int indexEnd = indexStart+ptsNb;
            reducedDivisor[i_day] = thrust::reduce(devDivisor.begin()+indexStart, devDivisor.begin()+indexEnd);
            reducedDividend[i_day] = thrust::reduce(devDividend.begin()+indexStart, devDividend.begin()+indexEnd);
        }*/
        thrust::reduce_by_key(keys.begin(), keys.end(), devDivisor.begin(), reducedKeys.begin(), reducedDivisor.begin());
        thrust::reduce_by_key(keys.begin(), keys.end(), devDividend.begin(), reducedKeys.begin(), reducedDividend.begin());

        #if DO_PROFILE
            stop = clock();   
            time = (float)(stop-start)/CLOCKS_PER_SEC*1000;
            fprintf(stderr, "Reduction: %f ms\n", time);

            start = clock();
        #endif //DO_PROFILE

        // Add to the resulting criteria
        thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(reducedDividend.begin(), reducedDivisor.begin(), resultingCriteria.begin())),
                         thrust::make_zip_iterator(thrust::make_tuple(reducedDividend.end(), reducedDivisor.end(), resultingCriteria.end())),
                         gpuAddToCriteriaS1grads(weights[i_ptor]));

        #if DO_PROFILE
            stop = clock();   
            time = (float)(stop-start)/CLOCKS_PER_SEC*1000;
            fprintf(stderr, "Final merging: %f ms\n", time);

            std::cout << "Press ENTER to continue... " << std::flush;
            std::cin.ignore( std::numeric_limits <std::streamsize> ::max(), '\n' );
        #endif //DO_PROFILE
    }

    // Copy to the final container
    thrust::copy(resultingCriteria.begin(), resultingCriteria.end(), criteriaValues.begin());


    #else // USE_THRUST

    // Error var
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return false;
    }

    // Get the meta data
    cudaPredictorsMetaDataStruct metaData;
    metaData.ptorsNb = (int)weights.size();
    if (metaData.ptorsNb>20)
    {
        printf("The number of predictors is >20. Please adapt the source code in asProcessorCuda::ProcessCriteria.");
        return false;
    }

    metaData.totPtsNb = 0;

    for (int i_ptor=0; i_ptor<metaData.ptorsNb; i_ptor++)
    {
        metaData.rowsNb[i_ptor] = rowsNb[i_ptor];
        metaData.colsNb[i_ptor] = colsNb[i_ptor];

        metaData.weights[i_ptor] = weights[i_ptor];
        metaData.ptsNb[i_ptor] = colsNb[i_ptor]*rowsNb[i_ptor];
        metaData.indexStart[i_ptor] = metaData.totPtsNb;
        metaData.indexEnd[i_ptor] = metaData.totPtsNb+metaData.ptsNb[i_ptor]-1;
        metaData.totPtsNb += colsNb[i_ptor]*rowsNb[i_ptor];

    }

    // Device copies of data
    float *devTargData, *devArchData, *devCriteriaValues;

    // Get data as arrays
    float* arrCriteriaValues = &criteriaValues[0];
    float* arrTargData;
    arrTargData = new float[metaData.totPtsNb];
    for (int i_ptor=0; i_ptor<metaData.ptorsNb; i_ptor++)
    {
        for (int i_pt=0; i_pt<metaData.ptsNb[i_ptor]; i_pt++)
        {
            arrTargData[metaData.indexStart[i_ptor] + i_pt] = vpTargData[i_ptor][i_pt];
        }
        //std::copy(vpTargData[i_ptor], vpTargData[i_ptor] + metaData.indexEnd[i_ptor], arrTargData + metaData.indexStart[i_ptor]); -> fails
    }
    float* arrArchData;
    arrArchData = new float[size*metaData.totPtsNb];
    for (int i_day=0; i_day<size; i_day++)
    {
        for (int i_ptor=0; i_ptor<metaData.ptorsNb; i_ptor++)
        {
            for (int i_pt=0; i_pt<metaData.ptsNb[i_ptor]; i_pt++)
            {
                arrArchData[i_day*metaData.totPtsNb + metaData.indexStart[i_ptor] + i_pt] = vvpArchData[i_day][i_ptor][i_pt];
            }
            //std::copy(vvpArchData[i_day][i_ptor], vvpArchData[i_day][i_ptor] + metaData.indexEnd[i_ptor], arrArchData + i_day*metaData.totPtsNb + metaData.indexStart[i_ptor]); -> fails
        }
    }

    // Alloc space for device copies of data
    int sizeTargData = metaData.totPtsNb*sizeof(float);
    cudaStatus = hipMalloc(&devTargData, sizeTargData);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for the target data!");
        delete[] arrTargData;
        hipFree(devTargData);
        return false;
    }

    int sizeArchData = size*metaData.totPtsNb*sizeof(float);
    cudaStatus = hipMalloc(&devArchData, sizeArchData);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for the archive data!");
        delete[] arrTargData;
        delete[] arrArchData;
        hipFree(devTargData);
        hipFree(devArchData);
        return false;
    }

    int sizeCriteriaValues = size*sizeof(float);
    cudaStatus = hipMalloc(&devCriteriaValues, sizeCriteriaValues);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for the criteria!");
        delete[] arrTargData;
        delete[] arrArchData;
        hipFree(devTargData);
        hipFree(devArchData);
        hipFree(devCriteriaValues);
        return false;
    }

    // Copy inputs to device
    cudaStatus = hipMemcpy(devCriteriaValues, arrCriteriaValues, sizeCriteriaValues, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for the criteria!");
        delete[] arrTargData;
        delete[] arrArchData;
        hipFree(devTargData);
        hipFree(devArchData);
        hipFree(devCriteriaValues);
        return false;
    }

    cudaStatus = hipMemcpy(devTargData, arrTargData, sizeTargData, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for the target data!");
        delete[] arrTargData;
        delete[] arrArchData;
        hipFree(devTargData);
        hipFree(devArchData);
        hipFree(devCriteriaValues);
        return false;
    }

    cudaStatus = hipMemcpy(devArchData, arrArchData, sizeArchData, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for the archive data!");
        delete[] arrTargData;
        delete[] arrArchData;
        hipFree(devTargData);
        hipFree(devArchData);
        hipFree(devCriteriaValues);
        return false;
    }

    // Launch kernel on GPU
    int threadsPerBlock = 512;
    int blocksNb = 1+size/threadsPerBlock;
    gpuPredictorCriteriaS1grads<<<blocksNb,threadsPerBlock>>>(devCriteriaValues, devTargData, devArchData, metaData, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        delete[] arrTargData;
        delete[] arrArchData;
        hipFree(devTargData);
        hipFree(devArchData);
        hipFree(devCriteriaValues);
        return false;
    }

    // hipDeviceSynchronize waits for the kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        delete[] arrTargData;
        delete[] arrArchData;
        hipFree(devTargData);
        hipFree(devArchData);
        hipFree(devCriteriaValues);
        return false;
    }

    // Copy result back to host
    cudaStatus = hipMemcpy(arrCriteriaValues, devCriteriaValues, sizeCriteriaValues, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for the results!");
        delete[] arrTargData;
        delete[] arrArchData;
        hipFree(devTargData);
        hipFree(devArchData);
        hipFree(devCriteriaValues);
        return false;
    }

    // Cleanup
    hipFree(devCriteriaValues);
    hipFree(devTargData);
    hipFree(devArchData);
    delete[] arrTargData;
    delete[] arrArchData;

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    /*cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }*/

    #endif // USE_THRUST

    return true;
}
