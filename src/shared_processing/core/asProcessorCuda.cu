#include "hip/hip_runtime.h"
/*
 * DO NOT ALTER OR REMOVE COPYRIGHT NOTICES OR THIS HEADER.
 *
 * The contents of this file are subject to the terms of the
 * Common Development and Distribution License (the "License").
 * You may not use this file except in compliance with the License.
 *
 * You can read the License at http://opensource.org/licenses/CDDL-1.0
 * See the License for the specific language governing permissions
 * and limitations under the License.
 *
 * When distributing Covered Code, include this CDDL Header Notice in
 * each file and include the License file (licence.txt). If applicable,
 * add the following below this CDDL Header, with the fields enclosed
 * by brackets [] replaced by your own identifying information:
 * "Portions Copyright [year] [name of copyright owner]"
 *
 * The Original Software is AtmoSwing.
 * The Original Software was developed at the University of Lausanne.
 * All Rights Reserved.
 *
 */

/*
 * Portions Copyright 2014-2015 Pascal Horton, Terranum.
 */

// Disable some MSVC warnings
#ifdef _MSC_VER
#pragma warning( disable : 4244 ) // C4244: conversion from 'unsigned __int64' to 'unsigned int', possible loss of data
#pragma warning( disable : 4267 ) // C4267: conversion from 'size_t' to 'int', possible loss of data
#endif


#include "asProcessorCuda.cuh"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>


__global__ void gpuPredictorCriteriaS1grads(float *criteria, const float *data, const int *indicesTarg,
                                            const int *indicesArch, const int *indexStart,
                                            const cudaPredictorsDataPropStruct dataProp, const int n_targ,
                                            const int n_cand, const int offset)
{
    int i_cand = offset + threadIdx.x + blockIdx.x * blockDim.x;
    if (i_cand < n_cand) {
        // Find the target index
        float meanNbCand = float(n_cand) / float(n_targ);
        int i_targ = floor(float(i_cand) / meanNbCand);

        if (i_targ < 0) {
            i_targ = 0;
        }

        if (i_targ >= n_targ) {
            i_targ = n_targ - 1;
        }

        // Check and correct
        if (i_cand < indexStart[i_targ]) {
            while (i_cand < indexStart[i_targ]) {
                i_targ--;

                if (i_targ < 0) {
                    printf("Device error: The target index is < 0 : i_targ = %d.\n", i_targ);
                    criteria[i_cand] = -9999;
                    return;
                }
            }
        }
        if (i_cand > indexStart[i_targ + 1]) // safe
        {
            while (i_cand > indexStart[i_targ + 1]) {
                i_targ++;

                if (i_targ >= n_targ) {
                    printf("Device error: The target index is >= n_targ : i_targ = %d (n_targ = %d)\n", i_targ, n_targ);
                    criteria[i_cand] = -9999;
                    return;
                }
            }
        }

        float criterion = 0;

        int targIndexBase = indicesTarg[i_targ] * dataProp.totPtsNb;
        int archIndexBase = indicesArch[i_cand] * dataProp.totPtsNb;

        for (int iPtor = 0; iPtor < dataProp.ptorsNb; iPtor++) {
            float dividend = 0, divisor = 0;
            int targIndex = targIndexBase + dataProp.indexStart[iPtor];
            int archIndex = archIndexBase + dataProp.indexStart[iPtor];

            for (int i = 0; i < dataProp.ptsNb[iPtor]; i++) {
                dividend += abs(data[targIndex] - data[archIndex]);
                divisor += max(abs(data[targIndex]), abs(data[archIndex]));

                targIndex++;
                archIndex++;
            }

            criterion += dataProp.weights[iPtor] * 100.0f * (dividend / divisor);
        }

        criteria[i_cand] = criterion;
    }
}


bool asProcessorCuda::ProcessCriteria(std::vector<std::vector<float *> > &data, std::vector<int> &indicesTarg,
                                      std::vector<std::vector<int> > &indicesArch,
                                      std::vector<std::vector<float> > &resultingCriteria, std::vector<int> &lengths,
                                      std::vector<int> &colsNb, std::vector<int> &rowsNb, std::vector<float> &weights)
{
    // Error var
    hipError_t cudaStatus;
    bool hasError = false;

    // Count the devices
    int devicesCount = 0;
    cudaStatus = hipGetDeviceCount(&devicesCount);
    if (cudaStatus != hipSuccess) {
        if (cudaStatus == hipErrorNoDevice) {
            fprintf(stderr, "hipGetDeviceCount failed! Do you have a CUDA-capable GPU installed?\n");
            return false;
        } else if (cudaStatus == hipErrorInsufficientDriver) {
            fprintf(stderr, "hipGetDeviceCount failed! No driver can be loaded to determine if any device exists.\n");
            return false;
        }

        fprintf(stderr, "hipGetDeviceCount failed! Do you have a CUDA-capable GPU installed?\n");
        return false;
    }

    // Get some info on the devices
    int bestDevice = 0;
    int memSize = 0;
    struct hipDeviceProp_t deviceProp;
    for (int i_dev = 0; i_dev < devicesCount; i_dev++) {
        cudaStatus = hipGetDeviceProperties(&deviceProp, i_dev);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGetDeviceProperties failed!\n");
            return false;
        }

        // Compare memory
        if (deviceProp.totalGlobalMem > memSize) {
            memSize = deviceProp.totalGlobalMem;
            bestDevice = i_dev;
        }
    }

    // Select the best device
    cudaStatus = hipSetDevice(bestDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!\n");
        return false;
    }

    // Get the data structure
    cudaPredictorsDataPropStruct dataProp;
    dataProp.ptorsNb = (int) weights.size();
    if (dataProp.ptorsNb > STRUCT_MAX_SIZE) {
        printf("The number of predictors is > %d. Please adapt the source code in asProcessorCuda::ProcessCriteria.\n",
               STRUCT_MAX_SIZE);
        return false;
    }

    dataProp.totPtsNb = 0;

    for (int iPtor = 0; iPtor < dataProp.ptorsNb; iPtor++) {
        dataProp.rowsNb[iPtor] = rowsNb[iPtor];
        dataProp.colsNb[iPtor] = colsNb[iPtor];
        dataProp.weights[iPtor] = weights[iPtor];
        dataProp.ptsNb[iPtor] = colsNb[iPtor] * rowsNb[iPtor];
        dataProp.indexStart[iPtor] = dataProp.totPtsNb;
        dataProp.totPtsNb += colsNb[iPtor] * rowsNb[iPtor];
    }

    // Sizes
    int lengthsSum = 0;
    std::vector<int> indexStart(lengths.size() + 1);
    for (int i_len = 0; i_len < lengths.size(); i_len++) {
        indexStart[i_len] = lengthsSum;
        lengthsSum += lengths[i_len];
    }
    indexStart[lengths.size()] = lengthsSum;
    int sizeData = dataProp.totPtsNb * data.size() * sizeof(float);
    int sizeCriteria = lengthsSum * sizeof(float);
    int sizeIndicesTarg = lengths.size() * sizeof(int);
    int sizeIndicesArch = lengthsSum * sizeof(int);
    int sizeIndexStart = (lengths.size() + 1) * sizeof(int); // + 1 relative to lengths

    // Create streams
    const int nStreams = 4; // no need to change
    //The number of threads per block should be a multiple of 32 threads, because this provides optimal computing efficiency and facilitates coalescing.
    const int threadsPerBlock = 1024; // no need to change
    // rowsNbPerStream must be dividable by nStreams and threadsPerBlock
    int rowsNbPerStream = ceil(float(lengthsSum) / float(nStreams * threadsPerBlock)) * threadsPerBlock;
    // Streams
    hipStream_t stream[nStreams];
    for (int i = 0; i < nStreams; i++) {
        hipStreamCreate(&stream[i]);
    }

    // Host and device pointers
    float *arrCriteria, *arrData;
    int *arrIndicesTarg = &indicesTarg[0];
    int *arrIndicesArch;
    int *arrIndexStart = &indexStart[0];
    float *devData, *devCriteria;
    int *devIndicesTarg, *devIndicesArch, *devIndexStart;

    // Alloc space for host copies of data

#if USE_PINNED_MEM

    // See http://devblogs.nvidia.com/parallelforall/how-optimize-data-transfers-cuda-cc/

    cudaStatus = hipHostAlloc((void **) &arrData, data.size() * dataProp.totPtsNb * sizeof(float),
                               hipHostMallocDefault);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipHostMalloc failed for the data!\n");
        hasError = true;
        goto cleanup;
    }

    cudaStatus = hipHostAlloc((void **) &arrIndicesArch, lengthsSum * sizeof(int), hipHostMallocDefault);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipHostMalloc failed for arrIndicesArch!\n");
        hasError = true;
        goto cleanup;
    }

    cudaStatus = hipHostAlloc((void **) &arrCriteria, lengthsSum * sizeof(float), hipHostMallocDefault);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipHostMalloc failed for arrCriteria!\n");
        hasError = true;
        goto cleanup;
    }

#else // USE_PINNED_MEM

    arrData = new float[data.size() * dataProp.totPtsNb];
    arrIndicesArch = new int[lengthsSum];
    arrCriteria = new float[lengthsSum];

#endif // USE_PINNED_MEM

    // Copy data in the new arrays
    for (int i_day = 0; i_day < data.size(); i_day++) {
        for (int iPtor = 0; iPtor < dataProp.ptorsNb; iPtor++) {
            for (int i_pt = 0; i_pt < dataProp.ptsNb[iPtor]; i_pt++) {
                arrData[i_day * dataProp.totPtsNb + dataProp.indexStart[iPtor] + i_pt] = data[i_day][iPtor][i_pt];
            }
            //std::copy(vvpArchData[i_day][iPtor], vvpArchData[i_day][iPtor] + dataProp.indexEnd[iPtor], arrArchData + i_day*dataProp.totPtsNb + dataProp.indexStart[iPtor]); -> fails
        }
    }

    for (int i_len = 0; i_len < lengths.size(); i_len++) {
        for (int j_len = 0; j_len < lengths[i_len]; j_len++) {
            arrIndicesArch[indexStart[i_len] + j_len] = indicesArch[i_len][j_len];
        }
    }

    // Alloc space for device copies of data
    cudaStatus = hipMalloc(&devData, sizeData);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for the data!\n");
        hasError = true;
        goto cleanup;
    }

    cudaStatus = hipMalloc(&devCriteria, sizeCriteria);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for the criteria!\n");
        hasError = true;
        goto cleanup;
    }

    cudaStatus = hipMalloc(&devIndicesTarg, sizeIndicesTarg);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for the target indices!\n");
        hasError = true;
        goto cleanup;
    }

    cudaStatus = hipMalloc(&devIndicesArch, sizeIndicesArch);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for the archive indices!\n");
        hasError = true;
        goto cleanup;
    }

    cudaStatus = hipMalloc(&devIndexStart, sizeIndexStart);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed for the archive indices!\n");
        hasError = true;
        goto cleanup;
    }

    /*
     * Asynchronous memcpy and processing. See:
     * https://github.com/parallel-forall/code-samples/blob/master/series/cuda-cpp/overlap-data-transfers/async.cu
     * http://devblogs.nvidia.com/parallelforall/how-overlap-data-transfers-cuda-cc/
     */

    // For the data, create its own stream
    cudaStatus = hipMemcpy(devData, arrData, sizeData, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for the data!\n");
        hasError = true;
        goto cleanup;
    }

    // No async for the indices as they don't use pinned memory
    cudaStatus = hipMemcpy(devIndicesTarg, arrIndicesTarg, sizeIndicesTarg, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for the target indices!\n");
        hasError = true;
        goto cleanup;
    }

    cudaStatus = hipMemcpy(devIndexStart, arrIndexStart, sizeIndexStart, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for the start indices!\n");
        hasError = true;
        goto cleanup;
    }

    // Make sure the data are copied
    //	hipDeviceSynchronize();

    // Copy archive indices to device
    for (int i = 0; i < nStreams; i++) {
        int offset = i * rowsNbPerStream;
        int length = 0;
        if (i < nStreams - 1) {
            length = rowsNbPerStream;
        } else {
            length = lengthsSum - offset; // Last slice
        }
        int streamBytes = length * sizeof(int);

        cudaStatus = hipMemcpyAsync(&devIndicesArch[offset], &arrIndicesArch[offset], streamBytes,
                                     hipMemcpyHostToDevice, stream[i]);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpyAsync failed for the archive data (stream %d/%d)!\n", i, nStreams);
            hasError = true;
            goto cleanup;
        }
    }

    // Launch kernel on GPU
    for (int i = 0; i < nStreams; i++) {
        int offset = i * rowsNbPerStream;
        int blocksNb = rowsNbPerStream / threadsPerBlock;
        int n_targ = lengths.size();
        int n_cand = lengthsSum;
        gpuPredictorCriteriaS1grads << < blocksNb, threadsPerBlock, 0, stream[i] >> >
                                                                       (devCriteria, devData, devIndicesTarg, devIndicesArch, devIndexStart, dataProp, n_targ, n_cand, offset);
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hasError = true;
        goto cleanup;
    }

    // Copy results back to host
    for (int i = 0; i < nStreams; i++) {
        int offset = i * rowsNbPerStream;
        int length = 0;
        if (i < nStreams - 1) {
            length = rowsNbPerStream;
        } else {
            length = lengthsSum - offset; // Last slice
        }
        int streamBytes = length * sizeof(float);

        cudaStatus = hipMemcpyAsync(&arrCriteria[offset], &devCriteria[offset], streamBytes, hipMemcpyDeviceToHost,
                                     stream[i]);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpyAsync failed for the results (stream %d/%d)!\n", i, nStreams);
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaStatus));
            hasError = true;
            goto cleanup;
        }
    }

    hipDeviceSynchronize();

    // Set the criteria values in the vector container
    for (int i_len = 0; i_len < lengths.size(); i_len++) {
        std::vector<float> tmpCrit(lengths[i_len]);

        for (int j_len = 0; j_len < lengths[i_len]; j_len++) {
            tmpCrit[j_len] = arrCriteria[indexStart[i_len] + j_len];
        }
        resultingCriteria[i_len] = tmpCrit;
    }

    // Cleanup
    cleanup:

    for (int i = 0; i < nStreams; i++) {
        hipStreamDestroy(stream[i]);
    }

    hipFree(devData);
    hipFree(devCriteria);
    hipFree(devIndicesTarg);
    hipFree(devIndicesArch);
    hipFree(devIndexStart);

#if USE_PINNED_MEM
    hipHostFree(arrData);
    hipHostFree(arrIndicesArch);
    hipHostFree(arrCriteria);
#else
    delete[] arrData;
    delete[] arrIndicesArch;
    delete[] arrCriteria;
#endif // USE_PINNED_MEM

    if (hasError)
        return false;

    return true;
}
