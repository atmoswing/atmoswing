#include "hip/hip_runtime.h"
/*
 * DO NOT ALTER OR REMOVE COPYRIGHT NOTICES OR THIS HEADER.
 *
 * The contents of this file are subject to the terms of the
 * Common Development and Distribution License (the "License").
 * You may not use this file except in compliance with the License.
 *
 * You can read the License at http://opensource.org/licenses/CDDL-1.0
 * See the License for the specific language governing permissions
 * and limitations under the License.
 *
 * When distributing Covered Code, include this CDDL Header Notice in
 * each file and include the License file (licence.txt). If applicable,
 * add the following below this CDDL Header, with the fields enclosed
 * by brackets [] replaced by your own identifying information:
 * "Portions Copyright [year] [name of copyright owner]"
 *
 * The Original Software is AtmoSwing.
 * The Original Software was developed at the University of Lausanne.
 * All Rights Reserved.
 *
 */

/*
 * Portions Copyright 2014-2015 Pascal Horton, Terranum.
 * Portions Copyright 2019 Pascal Horton, University of Bern.
 */

// Disable some MSVC warnings
#ifdef _MSC_VER
#pragma warning( disable : 4244 ) // C4244: conversion from 'unsigned __int64' to 'unsigned int', possible loss of data
#pragma warning( disable : 4267 ) // C4267: conversion from 'size_t' to 'int', possible loss of data
#endif

#include "asProcessorCuda.cuh"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>

// The number of threads per block should be a multiple of 32 threads, because this provides optimal computing
// efficiency and facilitates coalescing.
static const int blockSize = 1024;


// From https://riptutorial.com/cuda/example/22456/single-block-parallel-reduction-for-commutative-operator
__global__
void sumSingleBlock(int n, const float *a, float *out)
{
    int idx = threadIdx.x;
    float sum = 0;
    for (int i = idx; i < n; i += blockSize)
        sum += a[i];
    __shared__ float r[blockSize];
    r[idx] = sum;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) {
        if (idx < size)
            r[idx] += r[idx + size];
        __syncthreads();
    }
    if (idx == 0)
        *out = r[0];
}

// From https://devblogs.nvidia.com/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
__global__
void diff(int n, const float *x, const float *y, float *r)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < n; i += stride) {
        r[i] = x[i] - y[i];
    }
}

__global__
void maxAbs(int n, const float *x, const float *y, float *r)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < n; i += stride) {
        r[i] = fabs(x[i]);
        if (fabs(y[i]) > r[i]) {
            r[i] = fabs(y[i]);
        }
    }
}

__global__
void criteriaS1grads(int n, const float *x, const float *y, float *out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    __shared__ float diff[blockSize];
    __shared__ float amax[blockSize];

    for (int i = idx; i < n; i += stride) {
        float xi = x[i];
        float yi = y[i];

        float diffi = xi - yi;
        float amaxi = fabs(xi);
        if (fabs(yi) > amaxi) {
            amaxi = fabs(yi);
        }

        diff[i] = diffi;
        amax[i] = amaxi;
    }
    __syncthreads();

    float sumDiff = 0;
    float sumMax = 0;
    for (int i = idx; i < n; i += blockSize) {
        sumDiff += fabs(diff[i]);
        sumMax += amax[i];
    }

    __shared__ float rDiff[blockSize];
    __shared__ float rMax[blockSize];
    rDiff[idx] = sumDiff;
    rMax[idx] = sumMax;
    __syncthreads();

    for (int size = blockSize / 2; size > 0; size /= 2) {
        if (idx < size) {
            rDiff[idx] +=rDiff[idx + size];
            rMax[idx] +=rMax[idx + size];
        }
        __syncthreads();
    }
    if (idx == 0) {
        *out = 100.0f * (rDiff[0] / rMax[0]);

        if (rMax[0] == 0) {
            if (rDiff[0] == 0) {
                *out = 0;
            } else {
                *out = 200;
            }
        }
    }
}

bool asProcessorCuda::SelectBestDevice()
{
    hipError_t cudaStatus;
    bool showDeviceName = false;

    // Count the devices
    int devicesCount = 0;
    cudaStatus = hipGetDeviceCount(&devicesCount);
    if (cudaStatus != hipSuccess) {
        if (cudaStatus == hipErrorNoDevice) {
            printf("hipGetDeviceCount failed! Do you have a CUDA-capable GPU installed?\n");
            return false;
        } else if (cudaStatus == hipErrorInsufficientDriver) {
            printf("hipGetDeviceCount failed! No driver can be loaded to determine if any device exists.\n");
            return false;
        }

        printf("hipGetDeviceCount failed! Do you have a CUDA-capable GPU installed?\n");
        return false;
    }

    // Get some info on the devices
    int bestDevice = 0;
    int memSize = 0;
    struct hipDeviceProp_t deviceProps;
    for (int i_dev = 0; i_dev < devicesCount; i_dev++) {
        checkCudaErrors(hipGetDeviceProperties(&deviceProps, i_dev));
        if (showDeviceName) {
            printf("CUDA device [%s]\n", deviceProps.name);
        }

        // Compare memory
        if (deviceProps.totalGlobalMem > memSize) {
            memSize = deviceProps.totalGlobalMem;
            bestDevice = i_dev;
        }
    }

    // Select the best device
    checkCudaErrors(hipSetDevice(bestDevice));

    return true;
}

float *asProcessorCuda::MallocCudaData(int n)
{
    float *data;
    checkCudaErrors(hipMallocManaged(&data, n * sizeof(float)));

    return data;
}

void asProcessorCuda::FreeCudaData(float *data)
{
    checkCudaErrors(hipFree(data));
}

void asProcessorCuda::DeviceSynchronize()
{
    checkCudaErrors(hipDeviceSynchronize());
}

void asProcessorCuda::DeviceReset()
{
    hipDeviceReset();
}

bool asProcessorCuda::ProcessS1grads(float *out, const float *refData, const float *evalData, int rowsNb, int colsNb)
{
    int n = rowsNb * colsNb;

 

    int blocksNb = (n + blockSize - 1) / blockSize;

    if (blocksNb > 1) {
        printf("blocksNb > 1\n");
        return false;
    }


    bool m_checkNaNs = false;


    // Note here that the actual gradient data do not fill the entire data blocks,
    // but the rest being 0-filled, we can simplify the sum calculation !

    if (!m_checkNaNs) {
        criteriaS1grads<<<blocksNb, blockSize>>>(n, refData, evalData, out);
    } else {
        /*
        a2f refDataCorr = (!evalData.isNaN() && !refData.isNaN()).select(refData, 0);
        a2f evalDataCorr = (!evalData.isNaN() && !refData.isNaN()).select(evalData, 0);

        dividend = ((refDataCorr - evalDataCorr).abs()).sum();
        divisor = (refDataCorr.abs().max(evalDataCorr.abs())).sum();*/
    }

    return true;
}


__global__
void allPredictorsCriteriaS1grads(float *criteria, const float *data, const int *indicesTarg,
                                  const int *indicesArch, const cudaPredictorsDataPropStruct dataProp,
                                  const int n_cand, const int offset)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x + offset;
    int stride = blockDim.x * gridDim.x;

    for (int iCand = idx; iCand < n_cand; iCand += stride) {

        float criterion;
        int targIndexBase = indicesTarg[iCand] * dataProp.totPtsNb;
        int archIndexBase = indicesArch[iCand] * dataProp.totPtsNb;

        for (int iPtor = 0; iPtor < dataProp.ptorsNb; iPtor++) {
            int targIndex = targIndexBase + dataProp.indexStart[iPtor];
            int archIndex = archIndexBase + dataProp.indexStart[iPtor];



            int blocksNb = 1;

            __shared__ float r;

            switch (dataProp.criteria[iPtor]) {
                case S1grads:
                    criteriaS1grads< < <blocksNb, blockSize> > >(dataProp.ptsNb[iPtor], &data[targIndex], &data[archIndex], &r);
                default:
                    printf("Incorrect criteria provided.");
            }
            hipDeviceSynchronize();

            criterion += dataProp.weights[iPtor] * r;
        }

        criteria[iCand] = criterion;
    }

}

bool asProcessorCuda::ProcessCriteria(std::vector<std::vector<float *>> &data, std::vector<int> &indicesTarg,
                                      std::vector<std::vector<int>> &indicesArch,
                                      std::vector<std::vector<float>> &resultingCriteria,
                                      std::vector<int> &nbCandidates, const cudaPredictorsDataPropStruct &struc)
{

    // Sizes
    int nbCandidatesSum = 0;
    std::vector<int> indexStart(nbCandidates.size() + 1);
    for (int i = 0; i < nbCandidates.size(); i++) {
        indexStart[i] = nbCandidatesSum;
        nbCandidatesSum += nbCandidates[i];
    }
    indexStart[nbCandidates.size()] = nbCandidatesSum;

    // Blocks of threads
    int n_cand = nbCandidatesSum;
    int blocksNb = (n_cand + blockSize - 1) / blockSize;


#if USE_STREAMS
    // Create streams
    const int nStreams = 4; // no need to change
    // rowsNbPerStream must be dividable by nStreams and blockSize
    int rowsNbPerStream = ceil(float(nbCandidatesSum) / float(nStreams * blockSize)) * blockSize;
    // Streams
    hipStream_t stream[nStreams];
    for (int i = 0; i < nStreams; i++) {
        hipStreamCreate(&stream[i]);
    }
#endif

    // Data pointers
    float *arrData, *arrCriteria;
    int *arrIndicesTarg, *arrIndicesArch;

    // Alloc space for data
    checkCudaErrors(hipMallocManaged(&arrData, data.size() * struc.totPtsNb * sizeof(float)));
    checkCudaErrors(hipMallocManaged(&arrCriteria, nbCandidatesSum * sizeof(float)));
    checkCudaErrors(hipMallocManaged(&arrIndicesTarg, nbCandidatesSum * sizeof(int)));
    checkCudaErrors(hipMallocManaged(&arrIndicesArch, nbCandidatesSum * sizeof(int)));

    // Copy data in the new arrays
    for (int iDay = 0; iDay < data.size(); iDay++) {
        for (int iPtor = 0; iPtor < struc.ptorsNb; iPtor++) {
            for (int iPt = 0; iPt < struc.ptsNb[iPtor]; iPt++) {
                arrData[iDay * struc.totPtsNb + struc.indexStart[iPtor] + iPt] = data[iDay][iPtor][iPt];
            }
        }
    }

    for (int i = 0; i < indicesTarg.size(); i++) {
        for (int j = 0; j < nbCandidates[i]; j++) {
            arrIndicesArch[indexStart[i] + j] = indicesArch[i][j];
            arrIndicesTarg[indexStart[i] + j] = indicesTarg[i];
        }
    }

    // Launch kernel on GPU
#if USE_STREAMS
    for (int i = 0; i < nStreams; i++) {
        int offset = i * rowsNbPerStream;
        blocksNb = rowsNbPerStream / blockSize;
        gpuPredictorCriteriaS1grads<<<blocksNb, blockSize, 0, stream[i]>>>(arrCriteria, arrData, arrIndicesTarg, arrIndicesArch, struc, n_cand, offset);
    }
#else
    allPredictorsCriteriaS1grads< < <blocksNb, blockSize> > >(arrCriteria, arrData, arrIndicesTarg, arrIndicesArch, struc, n_cand, 0);
#endif

    // Check for any errors launching the kernel
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Set the criteria values in the vector container
    for (int i = 0; i < nbCandidates.size(); i++) {
        std::vector<float> tmpCrit(nbCandidates[i]);

        for (int j = 0; j < nbCandidates[i]; j++) {
            tmpCrit[j] = arrCriteria[indexStart[i] + j];
        }
        resultingCriteria[i] = tmpCrit;
    }

    // Cleanup

#if USE_STREAMS
    for (int i = 0; i< nStreams; i++) {
        hipStreamDestroy(stream[i]);
    }
#endif

    hipFree(arrData);
    hipFree(arrCriteria);
    hipFree(arrIndicesTarg);
    hipFree(arrIndicesArch);

    return false;
}

