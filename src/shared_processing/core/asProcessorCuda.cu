#include "hip/hip_runtime.h"
/*
 * DO NOT ALTER OR REMOVE COPYRIGHT NOTICES OR THIS HEADER.
 *
 * The contents of this file are subject to the terms of the
 * Common Development and Distribution License (the "License").
 * You may not use this file except in compliance with the License.
 *
 * You can read the License at http://opensource.org/licenses/CDDL-1.0
 * See the License for the specific language governing permissions
 * and limitations under the License.
 *
 * When distributing Covered Code, include this CDDL Header Notice in
 * each file and include the License file (licence.txt). If applicable,
 * add the following below this CDDL Header, with the fields enclosed
 * by brackets [] replaced by your own identifying information:
 * "Portions Copyright [year] [name of copyright owner]"
 *
 * The Original Software is AtmoSwing.
 * The Original Software was developed at the University of Lausanne.
 * All Rights Reserved.
 *
 */

/*
 * Portions Copyright 2014-2015 Pascal Horton, Terranum.
 * Portions Copyright 2019 Pascal Horton, University of Bern.
 */

// Disable some MSVC warnings
#ifdef _MSC_VER
#pragma warning( disable : 4244 ) // C4244: conversion from 'unsigned __int64' to 'unsigned int', possible loss of data
#pragma warning( disable : 4267 ) // C4267: conversion from 'size_t' to 'int', possible loss of data
#endif

#include "asProcessorCuda.cuh"
#include <stdio.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>

#define _TIME_CUDA true

// The number of threads per block should be a multiple of 32 threads, because this provides optimal computing
// efficiency and facilitates coalescing.
static const int blockSize = 64; // must be 64 <= blockSize <= 1024

__device__
void warpReduce64(volatile float *shared, int tid)
{
    shared[tid] += shared[tid + 32];
    shared[tid] += shared[tid + 16];
    shared[tid] += shared[tid + 8];
    shared[tid] += shared[tid + 4];
    shared[tid] += shared[tid + 2];
    shared[tid] += shared[tid + 1];
}

__device__
void warpReduce32(volatile float *shared, int tid)
{
    shared[tid] += shared[tid + 16];
    shared[tid] += shared[tid + 8];
    shared[tid] += shared[tid + 4];
    shared[tid] += shared[tid + 2];
    shared[tid] += shared[tid + 1];
}

__global__
void processS1grads(long candNb, int ptsNbtot, const float *data, const long *idxTarg, const long *idxArch, float w, float *out)
{
    const long blockId = gridDim.x * gridDim.y * blockIdx.z + blockIdx.y * gridDim.x + blockIdx.x;
    const int threadId = threadIdx.x;

    if (blockId < candNb) {
        long iTarg = idxTarg[blockId];
        long iArch = idxArch[blockId];

        extern __shared__ float mem[];
        float *diff = mem;
        float *amax = &diff[blockSize];

        float rdiff = 0;
        float rmax = 0;

        int nLoops = ceil(double(ptsNbtot) / blockSize);
        for (int i = 0; i < nLoops; ++i) {
            int nPts = blockSize;
            if (i == nLoops-1) {
                nPts = ptsNbtot - (i * blockSize);
            }

            // Process differences and get abs max
            if (threadId < nPts) {
                // Lookup data value
                float xi = data[iTarg * ptsNbtot + i * blockSize + threadId];
                float yi = data[iArch * ptsNbtot + i * blockSize + threadId];

                diff[threadId] = fabsf(xi - yi);
                amax[threadId] = fmaxf(fabsf(xi), fabsf(yi));
            } else {
                // Set rest of the block to 0
                diff[threadId] = 0;
                amax[threadId] = 0;
            }
            __syncthreads();

            // Process sum reduction
            for (unsigned int stride = blockSize / 2; stride > 32; stride /= 2) {
                if (threadId < stride) {
                    diff[threadId] += diff[threadId + stride];
                    amax[threadId] += amax[threadId + stride];
                }
                __syncthreads();
            }
            if (threadId < 32) {
                warpReduce64(diff, threadId);
                warpReduce64(amax, threadId);
            }
            __syncthreads();

            if (threadId == 0) {
                rdiff += diff[0];
                rmax += amax[0];
            }
        }
        __syncthreads();

        // Process final score
        if (threadId == 0) {
            float res = 0;

            if (rmax == 0) {
                res = 200;
            } else {
                res = 100.0f * (rdiff / rmax);
            }
            *(out + blockId) += res * w;
        }
    }
}

bool asProcessorCuda::ProcessCriteria(std::vector<std::vector<float *>> &data, std::vector<int> &indicesTarg,
                                      std::vector<std::vector<int>> &indicesArch,
                                      std::vector<std::vector<float>> &resultingCriteria,
                                      std::vector<int> &nbCandidates, std::vector<int> &colsNb,
                                      std::vector<int> &rowsNb, std::vector<float> &weights,
                                      std::vector<CudaCriteria> &criteria)
{
    int ptorsNb = weights.size();

#if _TIME_CUDA
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0.0f;
#endif

    // Sizes
    long candNb = 0;
    std::vector<long> indexStart(nbCandidates.size() + 1);
    for (int i = 0; i < nbCandidates.size(); i++) {
        indexStart[i] = candNb;
        candNb += nbCandidates[i];
    }
    indexStart[nbCandidates.size()] = candNb;

    // Alloc space for indices
#if _TIME_CUDA
    hipEventRecord(start);
#endif
    long *hIdxTarg, *dIdxTarg;
    hIdxTarg = (long *)malloc(candNb * sizeof(long));
    checkCudaErrors(hipMalloc((void **)&dIdxTarg, candNb * sizeof(long)));
    long *hIdxArch, *dIdxArch;
    hIdxArch = (long *)malloc(candNb * sizeof(long));
    checkCudaErrors(hipMalloc((void **)&dIdxArch, candNb * sizeof(long)));
#if _TIME_CUDA
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time to allocate IdxTarg and IdxArch:    %f\n", milliseconds);
#endif

#if _TIME_CUDA
    hipEventRecord(start);
#endif
    for (int i = 0; i < indicesTarg.size(); i++) {
        for (int j = 0; j < nbCandidates[i]; j++) {
            hIdxArch[indexStart[i] + j] = indicesArch[i][j];
            hIdxTarg[indexStart[i] + j] = indicesTarg[i];
        }
    }
#if _TIME_CUDA
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time to initialize IdxTarg and IdxArch:  %f\n", milliseconds);
#endif

    // Copy to device
#if _TIME_CUDA
    hipEventRecord(start);
#endif
    checkCudaErrors(hipMemcpy(dIdxTarg, hIdxTarg, candNb * sizeof(long), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dIdxArch, hIdxArch, candNb * sizeof(long), hipMemcpyHostToDevice));
#if _TIME_CUDA
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time to copy IdxTarg and IdxArch:        %f\n", milliseconds);
#endif

    // Alloc space for results
#if _TIME_CUDA
    hipEventRecord(start);
#endif
    float *hRes, *dRes;
    hRes = (float *)malloc(candNb * sizeof(float));
    checkCudaErrors(hipMalloc((void **)&dRes, candNb * sizeof(float)));
#if _TIME_CUDA
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time to allocate dRes:                   %f\n", milliseconds);
#endif

    // Init resulting array to 0s
#if _TIME_CUDA
    hipEventRecord(start);
#endif
    checkCudaErrors(hipMemset(dRes, 0, candNb * sizeof(float)));
#if _TIME_CUDA
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time to memset dRes:                     %f\n", milliseconds);
#endif

    // Get max predictor size
    long maxDataSize = 0;
    for (int iPtor = 0; iPtor < ptorsNb; iPtor++) {
        int ptsNb = colsNb[iPtor] * rowsNb[iPtor];
        long dataSize = data[iPtor].size() * ptsNb;
        if (dataSize > maxDataSize) {
            maxDataSize = dataSize;
        }
    }

    // Alloc space for data
#if _TIME_CUDA
    hipEventRecord(start);
#endif
    float *hData, *dData;
    hData = (float *)malloc(maxDataSize * sizeof(float));
    checkCudaErrors(hipMalloc((void **)&dData, maxDataSize * sizeof(float)));
#if _TIME_CUDA
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time to allocate dData:                  %f\n", milliseconds);
#endif

    // Loop over all predictors
    for (int iPtor = 0; iPtor < ptorsNb; iPtor++) {

        int ptsNb = colsNb[iPtor] * rowsNb[iPtor];
        float weight = weights[iPtor];
        long dataSize = data[iPtor].size() * ptsNb;

        // Copy data in the new arrays
#if _TIME_CUDA
        hipEventRecord(start);
#endif
        for (int iDay = 0; iDay < data[iPtor].size(); iDay++) {
            for (int iPt = 0; iPt < ptsNb; iPt++) {
                hData[iDay * ptsNb + iPt] = data[iPtor][iDay][iPt];
            }
        }
#if _TIME_CUDA
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("time to initialize hData:                %f\n", milliseconds);
#endif

        // Copy the data to the device
#if _TIME_CUDA
        hipEventRecord(start);
#endif
        checkCudaErrors(hipMemcpy(dData, hData, dataSize * sizeof(float), hipMemcpyHostToDevice));
#if _TIME_CUDA
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("time to copy data:                       %f\n", milliseconds);
#endif

        // Define block size (must be multiple of 32) and blocks nb
        int blocksNbXY = ceil(std::cbrt(candNb));
        int blocksNbZ = ceil((double)candNb / (blocksNbXY * blocksNbXY));
        dim3 blocksNb3D(blocksNbXY, blocksNbXY, blocksNbZ);

        // Launch kernel
#if _TIME_CUDA
        hipEventRecord(start);
#endif
        switch (criteria[iPtor]) {
            case S1grads:
                // 3rd <<< >>> argument is for the dynamically allocated shared memory
                processS1grads<<<blocksNb3D, blockSize, 2*blockSize*sizeof(float)>>>(candNb, ptsNb, dData, dIdxTarg, dIdxArch, weight, dRes);
                break;
            default:
                printf("Criteria not yet implemented on GPU.");
                return false;
        }

        // Check for any errors launching the kernel
        checkCudaErrors(hipGetLastError());

        checkCudaErrors(hipDeviceSynchronize());
#if _TIME_CUDA
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("time for kernel:                         %f\n", milliseconds);
#endif
    }

    // Copy the resulting array to the device
#if _TIME_CUDA
    hipEventRecord(start);
#endif
    checkCudaErrors(hipMemcpy(hRes, dRes, candNb * sizeof(float), hipMemcpyDeviceToHost));
#if _TIME_CUDA
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("time to copy results:                    %f\n", milliseconds);
#endif

    // Set the criteria values in the vector container
    for (int i = 0; i < nbCandidates.size(); i++) {
        std::vector<float> tmpCrit(nbCandidates[i]);

        for (int j = 0; j < nbCandidates[i]; j++) {
            tmpCrit[j] = hRes[indexStart[i] + j];
        }
        resultingCriteria[i] = tmpCrit;
    }

    free(hData);
    checkCudaErrors(hipFree(dData));
    free(hRes);
    checkCudaErrors(hipFree(dRes));
    free(hIdxTarg);
    checkCudaErrors(hipFree(dIdxTarg));
    free(hIdxArch);
    checkCudaErrors(hipFree(dIdxArch));

    return true;
}

bool asProcessorCuda::SelectBestDevice()
{
    hipError_t cudaStatus;
    bool showDeviceName = false;

    // Count the devices
    int devicesCount = 0;
    cudaStatus = hipGetDeviceCount(&devicesCount);
    if (cudaStatus != hipSuccess) {
        if (cudaStatus == hipErrorNoDevice) {
            printf("hipGetDeviceCount failed! Do you have a CUDA-capable GPU installed?\n");
            return false;
        } else if (cudaStatus == hipErrorInsufficientDriver) {
            printf("hipGetDeviceCount failed! No driver can be loaded to determine if any device exists.\n");
            return false;
        }

        printf("hipGetDeviceCount failed! Do you have a CUDA-capable GPU installed?\n");
        return false;
    }

    // Get some info on the devices
    int bestDevice = 0;
    int memSize = 0;
    struct hipDeviceProp_t deviceProps;
    for (int i_dev = 0; i_dev < devicesCount; i_dev++) {
        checkCudaErrors(hipGetDeviceProperties(&deviceProps, i_dev));
        if (showDeviceName) {
            printf("CUDA device [%s]\n", deviceProps.name);
        }

        // Compare memory
        if (deviceProps.totalGlobalMem > memSize) {
            memSize = deviceProps.totalGlobalMem;
            bestDevice = i_dev;
        }
    }

    // Select the best device
    checkCudaErrors(hipSetDevice(bestDevice));

    return true;
}

float *asProcessorCuda::MallocCudaData(int n)
{
    float *data;
    checkCudaErrors(hipMallocManaged(&data, n * sizeof(float)));

    return data;
}

void asProcessorCuda::FreeCudaData(float *data)
{
    checkCudaErrors(hipFree(data));
}

void asProcessorCuda::DeviceSynchronize()
{
    checkCudaErrors(hipDeviceSynchronize());
}

void asProcessorCuda::DeviceReset()
{
    hipDeviceReset();
}

