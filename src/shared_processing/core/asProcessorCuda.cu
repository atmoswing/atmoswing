#include "hip/hip_runtime.h"
/*
 * DO NOT ALTER OR REMOVE COPYRIGHT NOTICES OR THIS HEADER.
 *
 * The contents of this file are subject to the terms of the
 * Common Development and Distribution License (the "License").
 * You may not use this file except in compliance with the License.
 *
 * You can read the License at http://opensource.org/licenses/CDDL-1.0
 * See the License for the specific language governing permissions
 * and limitations under the License.
 *
 * When distributing Covered Code, include this CDDL Header Notice in
 * each file and include the License file (licence.txt). If applicable,
 * add the following below this CDDL Header, with the fields enclosed
 * by brackets [] replaced by your own identifying information:
 * "Portions Copyright [year] [name of copyright owner]"
 *
 * The Original Software is AtmoSwing.
 * The Original Software was developed at the University of Lausanne.
 * All Rights Reserved.
 *
 */

/*
 * Portions Copyright 2014-2015 Pascal Horton, Terranum.
 * Portions Copyright 2019 Pascal Horton, University of Bern.
 */

// Disable some MSVC warnings
#ifdef _MSC_VER
#pragma warning( disable : 4244 ) // C4244: conversion from 'unsigned __int64' to 'unsigned int', possible loss of data
#pragma warning( disable : 4267 ) // C4267: conversion from 'size_t' to 'int', possible loss of data
#endif

#include "asProcessorCuda.cuh"
#include <stdio.h>
#include <cmath>

#define FULL_MASK 0xffffffff

// The number of threads per block should be a multiple of 32 threads, because this provides optimal computing
// efficiency and facilitates coalescing.
static const int blockSize = 64; // must be 64 <= blockSize <= 1024

hipStream_t *g_streams = new hipStream_t[nStreams];

// From https://devblogs.nvidia.com/faster-parallel-reductions-kepler/
__inline__ __device__
float warpReduceSum(float val)
{
    for (int offset = 32 / 2; offset > 0; offset /= 2)
        val += __shfl_down_sync(FULL_MASK, val, offset);

    return val;
}

__global__
void processS1grads(const float *data, long ptorStart, int candNb, int ptsNbtot, int idxTarg, const int *idxArch,
    float w, float *out, int offset)
{
    const int blockId = gridDim.x * gridDim.y * blockIdx.z + blockIdx.y * gridDim.x + blockIdx.x;
    const int threadId = threadIdx.x;

    if (blockId < candNb) {
        int iTarg = idxTarg;
        int iArch = idxArch[offset + blockId];

        extern __shared__ float mem[];
        float *diff = mem;
        float *amax = &diff[blockSize];

        float rdiff = 0;
        float rmax = 0;

        int nLoops = ceil(double(ptsNbtot) / blockSize);
        for (int i = 0; i < nLoops; ++i) {
            int nPts = blockSize;
            if (i == nLoops-1) {
                nPts = ptsNbtot - (i * blockSize);
            }

            // Process differences and get abs max
            if (threadId < nPts) {
                // Lookup data value
                float xi = data[ptorStart + iTarg * ptsNbtot + i * blockSize + threadId];
                float yi = data[ptorStart + iArch * ptsNbtot + i * blockSize + threadId];

                diff[threadId] = fabsf(xi - yi);
                amax[threadId] = fmaxf(fabsf(xi), fabsf(yi));
            } else {
                // Set rest of the block to 0
                diff[threadId] = 0;
                amax[threadId] = 0;
            }
            __syncthreads();

            // Process sum reduction
            for (unsigned int stride = blockSize / 2; stride >= 32; stride /= 2) {
                if (threadId < stride) {
                    diff[threadId] += diff[threadId + stride];
                    amax[threadId] += amax[threadId + stride];
                }
                __syncthreads();
            }

            float ldiff = diff[threadId];
            float lamax = amax[threadId];
            __syncthreads();

            if (threadId < 32) {
                ldiff = warpReduceSum(ldiff);
                lamax = warpReduceSum(lamax);
            }
            __syncthreads();

            if (threadId == 0) {
                rdiff += ldiff;
                rmax += lamax;
            }
        }
        __syncthreads();

        // Process final score
        if (threadId == 0) {
            if (rmax == 0) {
                *(out + offset + blockId) += 200.0f * w;
            } else {
                *(out + offset + blockId) += 100.0f * (rdiff / rmax) * w;
            }
        }
    }
}

bool asProcessorCuda::ProcessCriteria(const float *dData, std::vector<long> ptorStart, int indexTarg, const int *indicesArch,
                                      float *dRes, int nbCandidates, std::vector<int> &colsNb, std::vector<int> &rowsNb,
                                      std::vector<float> &weights, std::vector<CudaCriteria> &criteria, int streamId, int offset)
{
    for (int iPtor = 0; iPtor < ptorStart.size(); iPtor++) {
        int ptsNb = colsNb[iPtor] * rowsNb[iPtor];

        // Define block size (must be multiple of 32) and blocks nb
        int blocksNbXY = ceil(std::cbrt(nbCandidates));
        int blocksNbZ = ceil((double)nbCandidates / (blocksNbXY * blocksNbXY));
        dim3 blocksNb3D(blocksNbXY, blocksNbXY, blocksNbZ);

        // Launch kernel
        switch (criteria[iPtor]) {
            case S1grads:
                // 3rd <<< >>> argument is for the dynamically allocated shared memory
                processS1grads<<<blocksNb3D, blockSize, 2 * blockSize * sizeof(float), g_streams[streamId]>>>
                     (dData, ptorStart[iPtor], nbCandidates, ptsNb, indexTarg, indicesArch, weights[iPtor], dRes, offset);
                break;
            default:
                printf("Criteria not yet implemented on GPU.");
                return false;
        }
    }

    return true;
}

bool asProcessorCuda::SelectBestDevice()
{
    hipError_t cudaStatus;
    bool showDeviceName = false;

    // Count the devices
    int devicesCount = 0;
    cudaStatus = hipGetDeviceCount(&devicesCount);
    if (cudaStatus != hipSuccess) {
        if (cudaStatus == hipErrorNoDevice) {
            printf("hipGetDeviceCount failed! Do you have a CUDA-capable GPU installed?\n");
            return false;
        } else if (cudaStatus == hipErrorInsufficientDriver) {
            printf("hipGetDeviceCount failed! No driver can be loaded to determine if any device exists.\n");
            return false;
        }

        printf("hipGetDeviceCount failed! Do you have a CUDA-capable GPU installed?\n");
        return false;
    }

    // Get some info on the devices
    int bestDevice = 0;
    int memSize = 0;
    struct hipDeviceProp_t deviceProps;
    for (int i_dev = 0; i_dev < devicesCount; i_dev++) {
        checkCudaErrors(hipGetDeviceProperties(&deviceProps, i_dev));
        if (showDeviceName) {
            printf("CUDA device [%s]\n", deviceProps.name);
        }

        // Compare memory
        if (deviceProps.totalGlobalMem > memSize) {
            memSize = deviceProps.totalGlobalMem;
            bestDevice = i_dev;
        }
    }

    // Select the best device
    checkCudaErrors(hipSetDevice(bestDevice));

    return true;
}

void asProcessorCuda::InitStreams()
{
    for (int i = 0; i < nStreams; i++)
        hipStreamCreate(&(g_streams[i]));
}

void asProcessorCuda::DestroyStreams()
{
    for (int i = 0; i < nStreams; i++)
        hipStreamDestroy(g_streams[i]);
}

void asProcessorCuda::CudaMalloc(int *&data, int length)
{
    checkCudaErrors(hipMalloc((void **)&data, length * sizeof(int)));
}

void asProcessorCuda::CudaMalloc(float *&data, long length)
{
    checkCudaErrors(hipMalloc((void **)&data, length * sizeof(float)));
}

void asProcessorCuda::CudaMemset0(float *data, long length)
{
    checkCudaErrors(hipMemset(data, 0, length * sizeof(float)));
}

void asProcessorCuda::CudaMemset0Async(float *data, long length, int streamId)
{
    checkCudaErrors(hipMemsetAsync(data, 0, length * sizeof(float), g_streams[streamId]));
}

void asProcessorCuda::CudaMemCopyToDevice(int *devData, int *hostData, int length)
{
    checkCudaErrors(hipMemcpy(devData, hostData, length * sizeof(int), hipMemcpyHostToDevice));
}

void asProcessorCuda::CudaMemCopyToDeviceAsync(int *devData, int *hostData, int length, int streamId)
{
    checkCudaErrors(hipMemcpyAsync(devData, hostData, length * sizeof(int), hipMemcpyHostToDevice, g_streams[streamId]));
}

void asProcessorCuda::CudaMemCopyToDevice(float *devData, float *hostData, long length)
{
    checkCudaErrors(hipMemcpy(devData, hostData, length * sizeof(float), hipMemcpyHostToDevice));
}

void asProcessorCuda::CudaMemCopyFromDevice(int *hostData, int *devData, int length)
{
    checkCudaErrors(hipMemcpy(hostData, devData, length * sizeof(int), hipMemcpyDeviceToHost));
}

void asProcessorCuda::CudaMemCopyFromDeviceAsync(int *hostData, int *devData, int length, int streamId)
{
    checkCudaErrors(hipMemcpyAsync(hostData, devData, length * sizeof(int), hipMemcpyDeviceToHost, g_streams[streamId]));
}

void asProcessorCuda::CudaMemCopyFromDevice(float *hostData, float *devData, long length)
{
    checkCudaErrors(hipMemcpy(hostData, devData, length * sizeof(float), hipMemcpyDeviceToHost));
}

void asProcessorCuda::CudaMemCopyFromDeviceAsync(float *hostData, float *devData, long length, int streamId)
{
    checkCudaErrors(hipMemcpyAsync(hostData, devData, length * sizeof(float), hipMemcpyDeviceToHost, g_streams[streamId]));
}

void asProcessorCuda::CudaFree(int *data)
{
    checkCudaErrors(hipFree(data));
}

void asProcessorCuda::CudaFree(float *data)
{
    checkCudaErrors(hipFree(data));
}

void asProcessorCuda::CudaGetLastError()
{
    checkCudaErrors(hipGetLastError());
}

void asProcessorCuda::DeviceSynchronize()
{
    checkCudaErrors(hipDeviceSynchronize());
}

void asProcessorCuda::StreamSynchronize(int streamId)
{
    checkCudaErrors(hipStreamSynchronize(g_streams[streamId]));
}

void asProcessorCuda::DeviceReset()
{
    hipDeviceReset();
}
